/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2]; 	/* values at time t */
       //oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       //newval[MAXPOINTS+2]; 	/* values at time (t+dt) */


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__  void update(float *vd, float *od, float *nd, int nsteps, int tpoints){
   int i, j = (blockIdx.x * 1024) + threadIdx.x + 1;
   float x, fac, k, tmp;

   fac = 2.0 * PI;
   k = j-1; 
   tmp = tpoints - 1;

   x = k/tmp;
   vd[j] = sin (fac * x);

   od[j] = vd[j];
   if( j == 1 ){
      printf("Updating all points for all time steps...\n");
   }
   __syncthreads();

   /* Update values for each time step */
   for (i = 1; i<= nsteps; i++) {
      /* Update points along line for this time step */

      /* global endpoints */
      if ((j == 1) || (j  == tpoints))
         nd[j] = 0.0;
      else{
         float dtime, c, dx, tau, sqtau;
         dtime = 0.3;
         c = 1.0;
         dx = 1.0;
         tau = (c * dtime / dx);
         sqtau = tau * tau;
         nd[j] = (2.0 * vd[j]) - od[j] + (sqtau *  (-2.0)*vd[j]);
      }
      __syncthreads();

      /* Update old values with new values */
      od[j] = vd[j];
      vd[j] = nd[j];
      __syncthreads();
   }

}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();
	printf("Initializing points on the line...\n");

   int size = (tpoints+1) * sizeof(float);
   float *vd, *od, *nd;

   hipMalloc(&vd, size);
   //cudaMemcpy(vd, values, size, cudaMemcpyHostToDevice);
   hipMalloc(&od, size);
   //cudaMemcpy(od, oldval, size, cudaMemcpyHostToDevice);
   hipMalloc(&nd, size);

   int threadPerBlock = 1024;
   int numBlocks = (tpoints % threadPerBlock) ? tpoints/threadPerBlock + 1 : tpoints/threadPerBlock;
	update<<<numBlocks, threadPerBlock>>>(vd, od, nd, nsteps, tpoints);
   hipDeviceSynchronize();
	printf("Printing final results...\n");

   hipMemcpy(values, vd, size, hipMemcpyDeviceToHost);
   hipFree(vd);
   hipFree(od);
   hipFree(nd);

	printfinal();
	printf("\nDone.\n\n");
	
	return 0;
}